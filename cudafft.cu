#include "luaT.h"
#include "THC/THC.h"
#include <hipfft/hipfft.h>

// extract fft(real to complex) and ifft(complex to real) from spectral-lib code 
// and register them to a new lib.

static int fft1d_r2c(lua_State *L) {
    THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");
    THCudaTensor *output = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");

    luaL_argcheck(L, input->nDimension == 2, 2, "input should be 2D real tensor [nLines x N]");
    luaL_argcheck(L, output->nDimension == 3, 2, "output should be 2D complex tensor [nLines x (N/2+1) x 2]");
    
    long nInputLines = input->size[0];
    long N = input->size[1];

    // argument check
    luaL_argcheck(L, output->size[0] == nInputLines, 0, "input and output should have the same number of lines");
    luaL_argcheck(L, (N % 2) == 0, 0, "N should be multiple of 2");
    luaL_argcheck(L, output->size[1] == N/2+1, 0, "output should be N/2+1");
    luaL_argcheck(L, output->size[2] == 2, 0, "output should be complex");
    luaL_argcheck(L, THCudaTensor_isContiguous(NULL,input), 2, "input must be contiguous");
    luaL_argcheck(L, THCudaTensor_isContiguous(NULL,output), 2, "output must be contiguous");
    

    // raw pointers 
    float *input_data = THCudaTensor_data(NULL,input);
    hipComplex *output_data = (hipComplex*)THCudaTensor_data(NULL,output);
    
    // execute FFT
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_R2C, nInputLines);
    hipfftExecR2C(plan, (hipfftReal*)input_data, (hipfftComplex*)output_data);

    // clean up
    hipfftDestroy(plan);

    return 0;
}

static int fft1d_c2r(lua_State *L) {
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *output = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");

	luaL_argcheck(L, output->nDimension == 2, 2, "output should be 2D real tensor [nLines x N]");
   	luaL_argcheck(L, input->nDimension == 3, 2, "input should be 2D complex tensor [nLines x (N/2+1) x 2]");
	
	long nInputLines = input->size[0];
	long N = output->size[1];

	// argument check
	luaL_argcheck(L, output->size[0] == nInputLines, 0, "input and output should have the same number of lines");
	luaL_argcheck(L, (N % 2) == 0, 0, "N should be multiple of 2");
	luaL_argcheck(L, input->size[1] == N/2+1, 0, "input should be N/2+1");
	luaL_argcheck(L, input->size[2] == 2, 0, "input should be complex");
	luaL_argcheck(L, THCudaTensor_isContiguous(NULL,input), 2, "input must be contiguous");
	luaL_argcheck(L, THCudaTensor_isContiguous(NULL,output), 2, "output must be contiguous");
	

	// raw pointers 
	float *output_data = THCudaTensor_data(NULL,output);
	hipComplex *input_data = (hipComplex*)THCudaTensor_data(NULL,input);
	
	// execute FFT
	hipfftHandle plan;
	hipfftPlan1d(&plan, N, HIPFFT_C2R, nInputLines);
	hipfftExecC2R(plan, (hipfftComplex*)input_data, (hipfftReal*)output_data);

	// clean up
	hipfftDestroy(plan);
	return 0;	
}

static const struct luaL_reg cudafft_init [] = {
    {"fft1d_r2c", fft1d_r2c},
    {"fft1d_c2r", fft1d_c2r},
    {NULL, NULL}
};

LUA_EXTERNC int luaopen_libcudafft(lua_State *L) {
    luaL_openlib(L, "cudafft", cudafft_init, 0);
    lua_pop(L,1);
    return 1;
}
